#include "hip/hip_runtime.h"
#include "definitions.hpp"
#include "interface.hpp"

#include "kernels/boundary_flux.hpp"
#include "kernels/pointwise_flux.hpp"
#include "kernels/riemann_flux.hpp"

template <typename num_t>
__global__ void pointwise_eulercartesian_2d(const num_t *q, num_t *flux_x1, num_t *flux_x2, const int nb_elem_x1, const int nb_elem_x2, const int nb_solpts_tot)
{
  const int ind = threadIdx.x + blockIdx.x * blockDim.x;
  const int nmax = nb_elem_x1 * nb_elem_x2 * nb_solpts_tot;
  const int stride = nmax;
  if(ind < nmax)
  {
    // Store variables and pointers to compute the fluxes 
    kernel_params<num_t,euler_state_2d> params(q, flux_x1, flux_x2, nullptr, ind, stride);

    // Call the pointwise flux kernel
    pointwise_eulercartesian_2d_kernel(params);
  }
}

template <typename num_t>
__global__ void riemann_eulercartesian_ausm_2d(const num_t *q_itf, num_t *flux_itf, const int nb_elem_x1, const int nb_elem_x2, const int nb_solpts, const int direction, const int nmax_x, const int nmax_y, const int nmax_z)
{
 const int ix = blockIdx.x * blockDim.x + threadIdx.x;
 const int iy = blockIdx.y * blockDim.y + threadIdx.y;
 const int iz = blockIdx.z * blockDim.z + threadIdx.z;

  if (ix<nmax_x && iy < nmax_y && iz < nmax_z)
  {
    const int nb_solpts_riem = 2 * nb_solpts;
    const int stride = nb_elem_x1 * nb_elem_x2 * nb_solpts_riem;
    const int array_shape[4] = {4, nb_elem_x2, nb_elem_x1, nb_solpts_riem};

    if (direction==0)
    {
      // Initialize left-hand side parameters
      const int indl = get_c_index(0, ix, iy, nb_solpts + iz, array_shape);
      kernel_params<num_t,euler_state_2d> params_l(q_itf, flux_itf, nullptr, nullptr, indl, stride);

      // Initialize right-hand-size parameters
      const int indr = get_c_index(0, ix, iy+1, iz, array_shape);
      kernel_params<num_t,euler_state_2d> params_r(q_itf, flux_itf, nullptr, nullptr, indr, stride);
    
      // Call Riemann kernel on the horizontal direction
      riemann_eulercartesian_ausm_2d_kernel(params_l, params_r, direction);
      
    }
    else if (direction==1)
    {
      // Initialize left-hand side parameters
      const int indl = get_c_index(0, ix, iy, nb_solpts + iz, array_shape);
      kernel_params<num_t,euler_state_2d> params_l(q_itf, nullptr, flux_itf, nullptr, indl, stride);

      // Initialize right-hand-size parameters
      const int indr = get_c_index(0, ix+1, iy, iz, array_shape);
      kernel_params<num_t,euler_state_2d> params_r(q_itf, nullptr, flux_itf, nullptr, indr, stride);
    
      // Call Riemann kernel on the horizontal direction
      riemann_eulercartesian_ausm_2d_kernel(params_l, params_r, direction);
    }
  }
}


template <typename num_t>
__global__ void boundary_eulercartesian_2d(const num_t *q_itf, num_t *flux_itf, const int nb_elem_x1, const int nb_elem_x2, const int nb_solpts, const int direction, const int nmax_x, const int nmax_y)
{
 const int ix = blockIdx.x * blockDim.x + threadIdx.x;
 const int iy = blockIdx.y * blockDim.y + threadIdx.y;
 
  if(ix < nmax_x && iy < nmax_y)
  {
    const int nb_solpts_riem = 2 * nb_solpts;
    const int stride = nb_elem_x1 * nb_elem_x2 * nb_solpts_riem;
    const int array_shape[4] = {4, nb_elem_x2, nb_elem_x1, nb_solpts_riem}; 

    if (direction==0) 
    {       
      // Left flux
      const int indl = get_c_index(0, ix, 0, iy, array_shape);
      kernel_params<num_t,euler_state_2d> params_l(q_itf, flux_itf, nullptr, nullptr, indl, stride);
      boundary_eulercartesian_2d_kernel(params_l, 0);

      // Right flux
      const int indr = get_c_index(0, ix, nb_elem_x1-1, nb_solpts + iy, array_shape);
      kernel_params<num_t,euler_state_2d> params_r(q_itf, flux_itf, nullptr, nullptr, indr, stride);
      boundary_eulercartesian_2d_kernel(params_r, 0);
    }

    if (direction==1) 
    {       
      // Bottom flux
      const int indb = get_c_index(0, 0, ix, iy, array_shape);
      kernel_params<num_t,euler_state_2d> params_b(q_itf, nullptr, flux_itf, nullptr, indb, stride);
      boundary_eulercartesian_2d_kernel(params_b, 1);

      // Top flux
      const int indt = get_c_index(0, nb_elem_x2-1, ix, nb_solpts + iy, array_shape);
      kernel_params<num_t,euler_state_2d> params_t(q_itf, nullptr, flux_itf, nullptr, indt, stride);
      boundary_eulercartesian_2d_kernel(params_t, 1);
    }
  }
}


// Explicit instantiations for each argument type (float, double or complex)
extern "C"
{
  void pointwise_eulercartesian_2d_double(const double *q, double *flux_x1, double *flux_x2, const int nb_elem_x1, const int nb_elem_x2, const int nb_solpts_tot)
  {
    const int num_blocks = (nb_elem_x1 * nb_elem_x2 * nb_solpts_tot + BLOCK_SIZE - 1) / BLOCK_SIZE;
    pointwise_eulercartesian_2d<double><<<num_blocks,BLOCK_SIZE>>>(q,flux_x1,flux_x2,nb_elem_x1,nb_elem_x2,nb_solpts_tot);
  }
  void pointwise_eulercartesian_2d_complex(const cuda::std::complex<double> *q, cuda::std::complex<double>  *flux_x1, cuda::std::complex<double>  *flux_x2, const int nb_elem_x1, const int nb_elem_x2, const int nb_solpts_tot)
  {
    const int num_blocks = (nb_elem_x1 * nb_elem_x2 * nb_solpts_tot + BLOCK_SIZE - 1) / BLOCK_SIZE;
    pointwise_eulercartesian_2d<cuda::std::complex<double>><<<num_blocks,BLOCK_SIZE>>>(q,flux_x1,flux_x2,nb_elem_x1,nb_elem_x2,nb_solpts_tot);
  }


  void riemann_eulercartesian_ausm_2d_double(const double *q_itf_x1, const double *q_itf_x2, double *f_itf_x1, double *f_itf_x2, const int nb_elem_x1, const int nb_elem_x2, const int nb_solpts)
  {
    int width, height, depth;

    // Call Riemann solver on the horizontal direction
    width = nb_elem_x2; 
    height = nb_elem_x1 - 1;
    depth = nb_solpts;

    dim3 threads_per_block (8, 8, 8);
    dim3 num_blocks1 ((width  + threads_per_block.x - 1) / threads_per_block.x,
                     (height + threads_per_block.y - 1) / threads_per_block.y,
                     (depth  + threads_per_block.z - 1) / threads_per_block.z);

    riemann_eulercartesian_ausm_2d<double><<<num_blocks1,threads_per_block>>>(q_itf_x1,f_itf_x1,nb_elem_x1,nb_elem_x2,nb_solpts,0,width,height,depth);


    // Call Riemann solver on the vertical direction
    width = nb_elem_x2 - 1; 
    height = nb_elem_x1;
    depth = nb_solpts;

    dim3 num_blocks2 ((width  + threads_per_block.x - 1) / threads_per_block.x,
                      (height + threads_per_block.y - 1) / threads_per_block.y,
                      (depth  + threads_per_block.z - 1) / threads_per_block.z);

    riemann_eulercartesian_ausm_2d<double><<<num_blocks2,threads_per_block>>>(q_itf_x2,f_itf_x2,nb_elem_x1,nb_elem_x2,nb_solpts,1,width,height,depth);


    // Set the boundary fluxes on the horizontal direction
    dim3 threads_per_block2 (16, 16);

    width = nb_elem_x2;
    height = nb_solpts;

    dim3 num_blocks3 ((width  + threads_per_block2.x - 1) / threads_per_block2.x,
                      (height + threads_per_block2.y - 1) / threads_per_block2.y);

    boundary_eulercartesian_2d<double><<<num_blocks3,threads_per_block2>>>(q_itf_x1,f_itf_x1,nb_elem_x1,nb_elem_x2,nb_solpts,0,width,height);

    width = nb_elem_x1;
    height = nb_solpts;

    dim3 num_blocks4 ((width  + threads_per_block2.x - 1) / threads_per_block2.x,
                      (height + threads_per_block2.y - 1) / threads_per_block2.y);

    boundary_eulercartesian_2d<double><<<num_blocks4,threads_per_block2>>>(q_itf_x2,f_itf_x2,nb_elem_x1,nb_elem_x2,nb_solpts,1,width,height);
  }

  void riemann_eulercartesian_ausm_2d_complex(const complex_t *q_itf_x1, const complex_t *q_itf_x2, complex_t *f_itf_x1, complex_t *f_itf_x2, const int nb_elem_x1, const int nb_elem_x2, const int nb_solpts)
  {
    int width, height, depth;

    // Call Riemann solver on the horizontal direction
    width = nb_elem_x2; 
    height = nb_elem_x1 - 1;
    depth = nb_solpts;

    dim3 threads_per_block (8, 8, 8);
    dim3 num_blocks1 ((width  + threads_per_block.x - 1) / threads_per_block.x,
                     (height + threads_per_block.y - 1) / threads_per_block.y,
                     (depth  + threads_per_block.z - 1) / threads_per_block.z);

    riemann_eulercartesian_ausm_2d<complex_t><<<num_blocks1,threads_per_block>>>(q_itf_x1,f_itf_x1,nb_elem_x1,nb_elem_x2,nb_solpts,0,width,height,depth);

    // Call Riemann solver on the vertical direction
    width = nb_elem_x2 - 1; 
    height = nb_elem_x1;
    depth = nb_solpts;

    dim3 num_blocks2 ((width  + threads_per_block.x - 1) / threads_per_block.x,
                      (height + threads_per_block.y - 1) / threads_per_block.y,
                      (depth  + threads_per_block.z - 1) / threads_per_block.z);

    riemann_eulercartesian_ausm_2d<complex_t><<<num_blocks2,threads_per_block>>>(q_itf_x2,f_itf_x2,nb_elem_x1,nb_elem_x2,nb_solpts,1,width,height,depth);


    // Set the boundary fluxes on the horizontal direction
    dim3 threads_per_block2 (16, 16);

    width = nb_elem_x2;
    height = nb_solpts;

    dim3 num_blocks3 ((width  + threads_per_block2.x - 1) / threads_per_block2.x,
                      (height + threads_per_block2.y - 1) / threads_per_block2.y);

    boundary_eulercartesian_2d<complex_t><<<num_blocks3,threads_per_block2>>>(q_itf_x1,f_itf_x1,nb_elem_x1,nb_elem_x2,nb_solpts,0,width,height);

    width = nb_elem_x1;
    height = nb_solpts;

    dim3 num_blocks4 ((width  + threads_per_block2.x - 1) / threads_per_block2.x,
                      (height + threads_per_block2.y - 1) / threads_per_block2.y);

    boundary_eulercartesian_2d<complex_t><<<num_blocks4,threads_per_block2>>>(q_itf_x2,f_itf_x2,nb_elem_x1,nb_elem_x2,nb_solpts,1,width,height);
  }

  
}

